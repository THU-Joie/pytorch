
#include <hip/hip_runtime.h>
// Counter-based block synchronization. Only meant to be used for
// debugging and validating synchronization. This should be replaced
// with cuda::barrier::arrive_and_wait as that should be more robust.

namespace block_sync {

using CounterType = unsigned int;
static constexpr CounterType COUNTER_TYPE_MAX = ~(CounterType)0;
__shared__ CounterType sync_counter;

__device__ void init() {
  const unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x +
      threadIdx.z * blockDim.x * blockDim.y;
  if (tid == 0) {
    sync_counter = 0;
  }
  __syncthreads();
}

// Emulate __syncthreads() with a synchronization counter
__device__ void sync() {
  unsigned int backoff = 8;
  const unsigned int backoff_max = 256;
  const unsigned int num_threads = blockDim.x * blockDim.y * blockDim.z;

  __threadfence_block();

  // Use counter range only up to a limit so that the next val won't
  // overflow.

  const auto counter_max = (COUNTER_TYPE_MAX / num_threads) * num_threads;
  const auto old = atomicInc(&sync_counter, counter_max - 1);

  const auto next = (old / num_threads) * num_threads + num_threads;

  auto local_sync_counter = *(volatile CounterType*)(&sync_counter);

  // sync_counter may wrap around, which means local_sync_counter
  // becomes smaller than old. In that case, it's guaranteed that all
  // threads have incremented the counter.
  while (local_sync_counter < next && old < local_sync_counter) {
    __nanosleep(backoff);
    if (backoff < backoff_max) {
      backoff *= 2;
    }
    local_sync_counter = *(volatile CounterType*)(&sync_counter);
  }
}

} // namespace block_sync
